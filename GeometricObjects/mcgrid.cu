
#include <hip/hip_runtime.h>
#ifdef WCUDA

#include <cuda_runtime.h>
#include "ray.cuh"
#include "ray.h"
#include <curand_kernel.h>
#include "CUDAhelpers.h"
#include "mcgrid.h"
#include "shaderec.h"

static __device__ bool inside_bb(const CUDAreal3 &p0, const CUDAreal3 &p1, const CUDAreal3 &point) {
	return
		point.x > p0.x && point.x < p1.x &&
		point.y > p0.y && point.y < p1.x &&
		point.z > p0.z && point.z < p1.z;
}

static __device__ CUDAreal clamp(CUDAreal value, CUDAreal a, CUDAreal b) {
	if (value < a) return a;
	if (value > b) return b;
	return value;
}

__device__ bool MCGridCUDA::shadow_hit(const rayCU& ray, CUDAreal& tmin) const
{
	return false;
}

__device__ bool MCGridCUDA::hit(const rayCU& ray, CUDAreal& tmin, ShadeRecCUDA& sr) const
{
	//Material* mat_ptr = sr.material_ptr;

	// if it's a kernel:
	/*int column = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;


	rayCU ray = rays[column + stride * row];*/

	//else:



	//MCGridCUDA grid = *gr;

	//int nx = grid.nx;
	//int ny = grid.ny;
	//int nz = grid.nz;


	CUDAreal ox = ray.o.x;
	CUDAreal oy = ray.o.y;
	CUDAreal oz = ray.o.z;
	CUDAreal dx = ray.d.x;
	CUDAreal dy = ray.d.y;
	CUDAreal dz = ray.d.z;
	CUDAreal x0 = p0.x;
	CUDAreal y0 = p0.y;
	CUDAreal z0 = p0.z;
	CUDAreal x1 = p1.x;
	CUDAreal y1 = p1.y;
	CUDAreal z1 = p1.z;
	CUDAreal tx_min, ty_min, tz_min;
	CUDAreal tx_max, ty_max, tz_max;
	// the following code includes modifications from Shirley and Morley (2003)

	CUDAreal a = 1.0 / dx;
	if (a >= 0) {
		tx_min = (x0 - ox) * a;
		tx_max = (x1 - ox) * a;
	}
	else {
		tx_min = (x1 - ox) * a;
		tx_max = (x0 - ox) * a;
	}

	CUDAreal b = 1.0 / dy;
	if (b >= 0) {
		ty_min = (y0 - oy) * b;
		ty_max = (y1 - oy) * b;
	}
	else {
		ty_min = (y1 - oy) * b;
		ty_max = (y0 - oy) * b;
	}

	CUDAreal c = 1.0 / dz;
	if (c >= 0) {
		tz_min = (z0 - oz) * c;
		tz_max = (z1 - oz) * c;
	}
	else {
		tz_min = (z1 - oz) * c;
		tz_max = (z0 - oz) * c;
	}

	CUDAreal t0, t1;

	if (tx_min > ty_min)
		t0 = tx_min;
	else
		t0 = ty_min;

	if (tz_min > t0)
		t0 = tz_min;

	if (tx_max < ty_max)
		t1 = tx_max;
	else
		t1 = ty_max;

	if (tz_max < t1)
		t1 = tz_max;

	if (t0 > t1)
		return (false);


	// initial cell coordinates

	int ix, iy, iz;

	if (inside_bb(p0, p1, ray.o)) {  			// does the ray start inside the grid?
		ix = clamp((ox - x0) * nx / (x1 - x0), 0, nx - 1);
		iy = clamp((oy - y0) * ny / (y1 - y0), 0, ny - 1);
		iz = clamp((oz - z0) * nz / (z1 - z0), 0, nz - 1);
	}
	else {
		CUDAreal3 p = ray.o + t0 * ray.d;  // initial hit point with grid's bounding box
		ix = clamp((p.x - x0) * nx / (x1 - x0), 0, nx - 1);
		iy = clamp((p.y - y0) * ny / (y1 - y0), 0, ny - 1);
		iz = clamp((p.z - z0) * nz / (z1 - z0), 0, nz - 1);
	}

	// ray parameter increments per cell in the x, y, and z directions

	CUDAreal dtx = (tx_max - tx_min) / nx;
	CUDAreal dty = (ty_max - ty_min) / ny;
	CUDAreal dtz = (tz_max - tz_min) / nz;

	CUDAreal 	tx_next, ty_next, tz_next;
	int 	ix_step, iy_step, iz_step;
	int 	ix_stop, iy_stop, iz_stop;

	if (dx > 0) {
		tx_next = tx_min + (ix + 1) * dtx;
		ix_step = +1;
		ix_stop = nx;
	}
	else {
		tx_next = tx_min + (nx - ix) * dtx;
		ix_step = -1;
		ix_stop = -1;
	}

	if (dx == 0.0) {
		tx_next = kHugeValueCUDA;
		ix_step = -1;
		ix_stop = -1;
	}


	if (dy > 0) {
		ty_next = ty_min + (iy + 1) * dty;
		iy_step = +1;
		iy_stop = ny;
	}
	else {
		ty_next = ty_min + (ny - iy) * dty;
		iy_step = -1;
		iy_stop = -1;
	}

	if (dy == 0.0) {
		ty_next = kHugeValueCUDA;
		iy_step = -1;
		iy_stop = -1;
	}

	if (dz > 0) {
		tz_next = tz_min + (iz + 1) * dtz;
		iz_step = +1;
		iz_stop = nz;
	}
	else {
		tz_next = tz_min + (nz - iz) * dtz;
		iz_step = -1;
		iz_stop = -1;
	}

	if (dz == 0.0) {
		tz_next = kHugeValueCUDA;
		iz_step = -1;
		iz_stop = -1;
	}

	//    if (tx_next < 0) tx_next = kHugeValueCUDA;
	//    if (ty_next < 0) ty_next = kHugeValueCUDA;
	//    if (tz_next < 0) tz_next = kHugeValueCUDA;



	// Test if there is a block face glued to the bounding box:

	int block_id = cells[ix + nx * iy + nx * ny * iz];
	CUDAreal3 block_p0 = __make_CUDAreal3(x0 + nx * BLOCKLENGTH_CUDA, y0 + ny * BLOCKLENGTH_CUDA, z0 + nz * BLOCKLENGTH_CUDA);
	if (block_id != 0) {
		real t_before = kHugeValueCUDA;

		real tx_min_pp = tx_next - dtx;
		real ty_min_pp = ty_next - dty;
		real tz_min_pp = tz_next - dtz;

		if (ix != 0 && ix != (nx - 1)) tx_min_pp = -kHugeValueCUDA;
		if (iy != 0 && iy != (ny - 1)) ty_min_pp = -kHugeValueCUDA;
		if (iz != 0 && iz != (nz - 1)) tz_min_pp = -kHugeValueCUDA;


		if (tx_min_pp > ty_min_pp && tx_min_pp > tz_min_pp) {
			(sr).normal = __make_CUDAreal3(-(CUDAreal)ix_step, 0, 0);
			//sr.hdir = ix_step > 0 ? ShadeRec::South : ShadeRec::North;
			t_before = tx_min_pp;
		}
		else if (ty_min_pp > tz_min_pp) {
			(sr).normal = __make_CUDAreal3(0, -(CUDAreal)iy_step, 0);
			//sr.hdir = iy_step > 0 ? ShadeRec::Bottom : ShadeRec::Top;
			t_before = ty_min_pp;

		}
		else {
			(sr).normal = __make_CUDAreal3(0, 0, -(CUDAreal)iz_step);
			//sr.hdir = iz_step > 0 ? ShadeRec::West : ShadeRec::East;
			t_before = tz_min_pp;

		}
		if (true /*block_ptr->block_hit(ray, block_p0, t_before, sr)*/) {
			tmin = t_before;


			return (true);
		}
	}



	// traverse the grid
	tmin = kHugeValueCUDA;
	CUDAreal t_before = kHugeValueCUDA;

	while (true) {
		//        MCBlock* block_ptr = cells[ix + nx * iy + nx * ny * iz];

		if (tx_next < ty_next && tx_next < tz_next) {
			//real tmin = tx_next - kEpsilonCUDACUDA;
			//Material* mptr = sr.material_ptr;
			(sr).normal = __make_CUDAreal3(-(CUDAreal)ix_step, 0, 0);
			//sr.hdir = ix_step > 0 ? ShadeRec::South : ShadeRec::North;
			//sr.t_Before = t_before;
			t_before = tx_next;
			tx_next += dtx;
			ix += ix_step;
			if (ix == ix_stop) {
				(sr).material = 0;
				return (false);
			}


			int block_ptr = cells[ix + nx * iy + nx * ny * iz];
			CUDAreal3 block_p0 = __make_CUDAreal3(x0 + nx * BLOCKLENGTH_CUDA, y0 + ny * BLOCKLENGTH_CUDA, z0 + nz * BLOCKLENGTH_CUDA);

			if (block_ptr /* && block_ptr->block_hit(ray, block_p0, t_before, sr)*/) {
				tmin = t_before;


				return (true);
			}
			//sr.material_ptr = mptr;

		}
		else {
			if (ty_next < tz_next) {
				//Material* mptr = sr.material_ptr;
				(sr).normal = __make_CUDAreal3(0.0, -(CUDAreal)iy_step, 0);
				//sr.hdir = iy_step > 0 ? ShadeRec::Bottom : ShadeRec::Top;
				//sr.t_Before = t_before;
				t_before = ty_next;
				ty_next += dty;
				iy += iy_step;
				if (iy == iy_stop) {
					//(*sr).material = mat_ptr;
					return (false);
				}

				int block_ptr = cells[ix + nx * iy + nx * ny * iz];
				CUDAreal3 block_p0 = __make_CUDAreal3(x0 + nx * BLOCKLENGTH_CUDA, y0 + ny * BLOCKLENGTH_CUDA, z0 + nz * BLOCKLENGTH_CUDA);


				if (block_ptr /*&& block_ptr->block_hit(ray, block_p0, t_before, sr)*/) {
					//material_ptr = object_ptr->get_material();
					tmin = t_before;
					//t = ty_next;
					return (true);
				}
				//sr.material_ptr = mptr;
				//mat_ptr

			}
			else {
				//Material* mptr = sr.material_ptr;
				(sr).normal = __make_CUDAreal3(0.0, 0.0, -(CUDAreal)iz_step);
				//sr.hdir = iz_step > 0 ? ShadeRec::West : ShadeRec::East;
				//sr.t_Before = t_before;
				t_before = tz_next;
				tz_next += dtz;
				iz += iz_step;
				if (iz == iz_stop) {
					//sr.material_ptr = mat_ptr;
					return (false);
				}

				int block_ptr = cells[ix + nx * iy + nx * ny * iz];
				CUDAreal3 block_p0 = __make_CUDAreal3(x0 + nx * BLOCKLENGTH_CUDA, y0 + ny * BLOCKLENGTH_CUDA, z0 + nz * BLOCKLENGTH_CUDA);


				//material_ptr = sr.material_ptr;
				if (block_ptr/* && block_ptr->block_hit(ray, block_p0, t_before, sr)*/) {
					//material_ptr = object_ptr->get_material();
					//sr.material_ptr = material_ptr;
					tmin = t_before;
					// t = tz_next;
					return (true);
				}
				//sr.material_ptr = mptr;

			}
		}
	}
}


#endif // WCUDA
